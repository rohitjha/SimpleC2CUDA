#include "add_kernel.hu"
 #include<stdio.h>
int main(){
  int a[3][3];
  int b[3][3];
  int c[3][3];
  int i,j;
  printf("Enter the First matrix->");
  for(i=0;i<3;i++)
      for(j=0;j<3;j++)
           scanf("%d",&a[i][j]);
  printf("\nEnter the Second matrix->");
  for(i=0;i<3;i++)
      for(j=0;j<3;j++)
           scanf("%d",&b[i][j]);
  printf("\nThe First matrix is\n");
  for(i=0;i<3;i++){
      printf("\n");
      for(j=0;j<3;j++)
           printf("%d\t",a[i][j]);
  }
  printf("\nThe Second matrix is\n");
  for(i=0;i<3;i++){
      printf("\n");
      for(j=0;j<3;j++)
      printf("%d\t",b[i][j]);
   }

int *dev_a;
int *dev_b;
int *dev_c;

hipMalloc((void **) &dev_a, (3) *(3) * sizeof(int);
hipMalloc((void **) &dev_b, (3) *(3) * sizeof(int);
hipMalloc((void **) &dev_c, (3) *(3) * sizeof(int);

hipMemcpy(dev_a, a, (3) * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, (3) * sizeof(int), hipMemcpyHostToDevice);

{
	dim3 k0_dimBlock(3, 3);
	dim3 k0_dimGrid(1, 1);
	kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
}

hipMemcpy(c, dev_c, (3) * sizeof(int), hipMemcpyDeviceToHost);

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);

   printf("\nThe Addition of two matrix is\n");
   for(i=0;i<3;i++){
       printf("\n");
       for(j=0;j<3;j++)
            printf("%d\t",c[i][j]);
   }
   return 0;
}
