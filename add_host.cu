#include "hip/hip_runtime.h"
#include "add_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int main()
{
    int a[500][500];
    int b[500][500];
    int c[500][500];
    int i, j;
    double total_time;
    clock_t start, end;
  
    srand(time(NULL));
    for(i = 0; i < 500; i++)
        for(j = 0; j < 500; j++)
        {
            a[i][j] = rand();
            b[i][j] = rand();
        }
  
    start = clock();

    
int *dev_a;
int *dev_b;
int *dev_c;

hipMalloc((void **) &dev_a, (500) *(500) * sizeof(int);
hipMalloc((void **) &dev_b, (500) *(500) * sizeof(int);
hipMalloc((void **) &dev_c, (500) *(500) * sizeof(int);

hipMemcpy(dev_a, a, (500) * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, (500) * sizeof(int), hipMemcpyHostToDevice);

{
	dim3 k0_dimBlock(32, 32);
	dim3 k0_dimGrid(16, 16);
	kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
}

hipMemcpy(c, dev_c, (500) * sizeof(int), hipMemcpyDeviceToHost);

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);

    
    end = clock();//time count stops 
    printf("\nThe sum of the two matrices is\n");
    for(i=0;i<500;i++){
        printf("\n");
        for(j=0;j<500;j++)
            printf("%d\t",c[i][j]);
    }
    total_time = (double)(end - start)*1000 / CLOCKS_PER_SEC;//calulate total time in milliseconds
    printf("\nTime taken: %f milliseconds\n", total_time);
    return 0;
}
