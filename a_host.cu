#include "hip/hip_runtime.h"
#include "a_kernel.hu"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
int main()
{
int a[10000];
int b[10000];
int c[10000];
int i;
double total_time;
clock_t start, end;
srand(time(NULL));
for(i = 0; i < 10000; i++)
{
    a[i] = rand();
    b[i] = rand();
}

start = clock();//time count starts 

int *dev_a;
int *dev_b;
int *dev_c;

hipMalloc((void **) &dev_a, (10000) * sizeof(int);
hipMalloc((void **) &dev_b, (10000) * sizeof(int);
hipMalloc((void **) &dev_c, (10000) * sizeof(int);

hipMemcpy(dev_a, a, (10000) * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, (10000) * sizeof(int), hipMemcpyHostToDevice);

{
	dim3 k0_dimBlock(32);
	dim3 k0_dimGrid(313);
	kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
}

hipMemcpy(c, dev_c, (10000) * sizeof(int), hipMemcpyDeviceToHost);

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);


end = clock();//time count stops 
total_time = (double)(end - start)*1000 / CLOCKS_PER_SEC;//calulate total time in milliseconds
printf("\nTime taken: %f milliseconds\n", total_time);

for(i=0;i<10000;i++)
    printf("%d\n",c[i]);
return 0;
}
