#include "hip/hip_runtime.h"
#include "a_kernel.hu"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
int main()
{
int a[]={1,2,3,4};
int b[]={1,2,3,4};
int c[4];
int i;
double total_time;
 clock_t start, end;
start = clock();//time count starts 

int *dev_a;
int *dev_b;
int *dev_c;

hipMalloc((void **) &dev_a, (4) * sizeof(int);
hipMalloc((void **) &dev_b, (4) * sizeof(int);
hipMalloc((void **) &dev_c, (4) * sizeof(int);

hipMemcpy(dev_a, a, (4) * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_b, b, (4) * sizeof(int), hipMemcpyHostToDevice);

{
	dim3 k0_dimBlock(4);
	dim3 k0_dimGrid(1);
	kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
}

hipMemcpy(c, dev_c, (4) * sizeof(int), hipMemcpyDeviceToHost);

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);


end = clock();//time count stops 
 total_time = ((double) (end - start));//calulate total time
 printf("\nTime taken is: %f", total_time);

for(i=0;i<4;i++)
printf("%d\n",c[i]);
return 0;
}
