#include "hip/hip_runtime.h"
#include "a_kernel.hu"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
int main()
{
int a[]={1,2,3,4};
int b[]={1,2,3,4};
int c[4];
int i;
double total_time;
 clock_t start, end;
start = clock();//time count starts 
int *dev_a;
int *dev_b;
int *dev_c;
hipMalloc((void **) &dev_a, (4) * sizeof(int);
hipMalloc((void **) &dev_b, (4) * sizeof(int);
hipMalloc((void **) &dev_c, (4) * sizeof(int);
hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);


end = clock();//time count stops 
 total_time = ((double) (end - start));//calulate total time
 printf("\nTime taken is: %f", total_time);

for(i=0;i<4;i++)
printf("%d\n",c[i]);
return 0;
}
