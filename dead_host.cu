#include "dead_kernel.hu"
#include <stdlib.h>

int main()
{
	int a[1000];
    int b[1000];

	for (int i = 0; i < 1000; ++i)
		a[i] = i;

int *dev_a;
int *dev_b;

hipMalloc((void **) &dev_a, (1000) * sizeof(int);
hipMalloc((void **) &dev_b, (1000) * sizeof(int);

hipMemcpy(dev_a, a, (1000) * sizeof(int), hipMemcpyHostToDevice);

{
	dim3 k0_dimBlock(32);
	dim3 k0_dimGrid(32);
	kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b);
}

hipMemcpy(b, dev_b, (1000) * sizeof(int), hipMemcpyDeviceToHost);

hipFree(dev_a);
hipFree(dev_b);

	for (int i = 0; i < 1000; ++i)
		if (b[i] != a[i])
			return EXIT_FAILURE;

	return EXIT_SUCCESS;
}
