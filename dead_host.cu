#include "hip/hip_runtime.h"
#include "dead_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int main()
{
	int a[10000];
    int b[10000];

	for (int i = 0; i < 10000; ++i)
		a[i] = i;

    clock_t start = clock();
    
int *dev_a;
int *dev_b;

hipMalloc((void **) &dev_a, (10000) * sizeof(int);
hipMalloc((void **) &dev_b, (10000) * sizeof(int);

hipMemcpy(dev_a, a, (10000) * sizeof(int), hipMemcpyHostToDevice);

{
	dim3 k0_dimBlock(32);
	dim3 k0_dimGrid(313);
	kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b);
}

hipMemcpy(b, dev_b, (10000) * sizeof(int), hipMemcpyDeviceToHost);

hipFree(dev_a);
hipFree(dev_b);

	
    clock_t end = clock();
    double total_time = (double)(end - start)*1000 / CLOCKS_PER_SEC;
    printf ("\nTotal time is: %f milliseconds\n", total_time);

    for (int i = 0; i < 10000; ++i)
		if (b[i] != a[i])
			return EXIT_FAILURE;

	return EXIT_SUCCESS;
}
