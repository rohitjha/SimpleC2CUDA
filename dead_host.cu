#include "dead_kernel.hu"
#include <stdlib.h>

int main()
{
	int a[1000];
    int b[1000];

	for (int i = 0; i < 1000; ++i)
		a[i] = i;
int *dev_a;
int *dev_b;
hipMalloc((void **) &dev_a, (1000) * sizeof(int);
hipMalloc((void **) &dev_b, (1000) * sizeof(int);
hipFree(dev_a);
hipFree(dev_b);

	for (int i = 0; i < 1000; ++i)
		if (b[i] != a[i])
			return EXIT_FAILURE;

	return EXIT_SUCCESS;
}
