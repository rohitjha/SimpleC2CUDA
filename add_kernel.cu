#include "hip/hip_runtime.h"
#include "add_kernel.hu"

__global__ void kernel0(int *a, int *b, int *c)
{
	int b0 = blockIdx.y;
	int b1 = blockIdx.x;
	int t0 = threadIdx.y;
	int t1 = threadIdx.x;
	{
            c[t0*3+t1]=a[t0*3+t1]+b[t0*3+t1];


	}
}
