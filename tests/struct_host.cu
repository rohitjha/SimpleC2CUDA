#include <assert.h>
#include <stdio.h>
#include "struct_kernel.hu"
#include <stdlib.h>

struct s {
	int c[10][10];
};

int main()
{
	struct s a[10][10], b[10][10];

	for (int i = 0; i < 10; ++i)
		for (int j = 0; j < 10; ++j)
			for (int k = 0; k < 10; ++k)
				for (int l = 0; l < 10; ++l)
					a[i][j].c[k][l] = i + j + k + l;
	#define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
	{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	  struct s *dev_b;
	  
	  cudaCheckReturn(hipMalloc((void **) &dev_b, (10) * (10) * sizeof(struct s)));
	  
	  {
	    dim3 k0_dimBlock(4, 4, 10);
	    dim3 k0_dimGrid(1, 1);
	    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_b);
	    cudaCheckKernel();
	  }
	  
	  cudaCheckReturn(hipMemcpy(b, dev_b, (10) * (10) * sizeof(struct s), hipMemcpyDeviceToHost));
	  cudaCheckReturn(hipFree(dev_b));
	}
	for (int i = 0; i < 10; ++i)
		for (int j = 0; j < 10; ++j)
			for (int k = 0; k < 10; ++k)
				for (int l = 0; l < 10; ++l)
					if (b[i][j].c[k][l] != a[i][j].c[k][l])
						return EXIT_FAILURE;

	return EXIT_SUCCESS;
}
