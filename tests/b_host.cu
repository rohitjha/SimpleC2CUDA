#include <assert.h>
#include <stdio.h>
#include "b_kernel.hu"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
int main()
{
int a[]={1,2,3,4,5,6};
int b[]={1,2,3,4,5,6};
int c[4];
int i;

#define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

  int *dev_a;
  int *dev_b;
  int *dev_c;
  
  cudaCheckReturn(hipMalloc((void **) &dev_a, (4) * sizeof(int)));
  cudaCheckReturn(hipMalloc((void **) &dev_b, (4) * sizeof(int)));
  cudaCheckReturn(hipMalloc((void **) &dev_c, (4) * sizeof(int)));
  
  cudaCheckReturn(hipMemcpy(dev_a, a, (4) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheckReturn(hipMemcpy(dev_b, b, (4) * sizeof(int), hipMemcpyHostToDevice));
  {
    dim3 k0_dimBlock(4);
    dim3 k0_dimGrid(1);
    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
    cudaCheckKernel();
  }
  
  cudaCheckReturn(hipMemcpy(c, dev_c, (4) * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheckReturn(hipFree(dev_a));
  cudaCheckReturn(hipFree(dev_b));
  cudaCheckReturn(hipFree(dev_c));
}


for(i=0;i<4;i++)
printf("%d\n",c[i]);
return 0;
}
