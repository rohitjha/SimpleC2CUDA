#include <assert.h>
#include <stdio.h>
#include "call_kernel.hu"
#include <stdlib.h>

void copy_summary(int b[1000], int a[1000], int pos)
{
	b[pos] = 0;
	int c = a[pos];
}

#ifdef pencil_access
__attribute__((pencil_access(copy_summary)))
#endif
void copy(int b[1000], int a[1000], int pos);

int main()
{
	int a[1000], b[1000];

	for (int i = 0; i < 1000; ++i)
		a[i] = i;
	#define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
	{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	  int *dev_a;
	  int *dev_b;
	  
	  cudaCheckReturn(hipMalloc((void **) &dev_a, (1000) * sizeof(int)));
	  cudaCheckReturn(hipMalloc((void **) &dev_b, (1000) * sizeof(int)));
	  
	  cudaCheckReturn(hipMemcpy(dev_a, a, (1000) * sizeof(int), hipMemcpyHostToDevice));
	  {
	    dim3 k0_dimBlock(32);
	    dim3 k0_dimGrid(32);
	    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b);
	    cudaCheckKernel();
	  }
	  
	  cudaCheckReturn(hipMemcpy(b, dev_b, (1000) * sizeof(int), hipMemcpyDeviceToHost));
	  cudaCheckReturn(hipFree(dev_a));
	  cudaCheckReturn(hipFree(dev_b));
	}
	for (int i = 0; i < 1000; ++i)
		if (b[i] != a[i])
			return EXIT_FAILURE;

	return EXIT_SUCCESS;
}
