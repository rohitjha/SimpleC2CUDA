#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "a_kernel.hu"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
int main()
{
int a[]={1,2,3,4};
int b[]={1,2,3,4};
int c[4];
int i;
double total_time;
 clock_t start, end;
start = clock();//time count starts 
#define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

  int *dev_a;
  int *dev_b;
  int *dev_c;
  
  cudaCheckReturn(hipMalloc((void **) &dev_a, (4) * sizeof(int))); //memory allocation in host
  cudaCheckReturn(hipMalloc((void **) &dev_b, (4) * sizeof(int)));
  cudaCheckReturn(hipMalloc((void **) &dev_c, (4) * sizeof(int)));
  
  cudaCheckReturn(hipMemcpy(dev_a, a, (4) * sizeof(int), hipMemcpyHostToDevice)); //memory allocation in device
  cudaCheckReturn(hipMemcpy(dev_b, b, (4) * sizeof(int), hipMemcpyHostToDevice));
  {
    dim3 k0_dimBlock(4); //4 is no. of blocks 
    dim3 k0_dimGrid(1); //each block has 1 thread
    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
    cudaCheckKernel();
  }
  
  cudaCheckReturn(hipMemcpy(c, dev_c, (4) * sizeof(int), hipMemcpyDeviceToHost)); //copy result to host
  cudaCheckReturn(hipFree(dev_a)); //free  memory in device
  cudaCheckReturn(hipFree(dev_b));
  cudaCheckReturn(hipFree(dev_c));
}

end = clock();//time count stops 
 total_time = ((double) (end - start));//calulate total time
 printf("\nTime taken is: %f", total_time);

for(i=0;i<4;i++)
printf("%d\n",c[i]);
return 0;
}
