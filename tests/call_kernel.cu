#include "hip/hip_runtime.h"
#include "call_kernel.hu"
__global__ void kernel0(int *a, int *b)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    if (32 * b0 + t0 <= 999)
      copy(b, a, (32 * b0 + t0));
}
