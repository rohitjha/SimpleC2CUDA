#include "hip/hip_runtime.h"
#include "add_kernel.hu"
__global__ void kernel0(int *a, int *b, int *c)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    c[t0 * 3 + t1] = (a[t0 * 3 + t1] + b[t0 * 3 + t1]);
}
