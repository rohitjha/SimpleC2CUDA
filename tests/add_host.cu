#include <assert.h>
#include <stdio.h>
#include "add_kernel.hu"
 #include<stdio.h>
int main(){
  int a[3][3],b[3][3],c[3][3],i,j;
  printf("Enter the First matrix->");
  for(i=0;i<3;i++)
      for(j=0;j<3;j++)
           scanf("%d",&a[i][j]);
  printf("\nEnter the Second matrix->");
  for(i=0;i<3;i++)
      for(j=0;j<3;j++)
           scanf("%d",&b[i][j]);
  printf("\nThe First matrix is\n");
  for(i=0;i<3;i++){
      printf("\n");
      for(j=0;j<3;j++)
           printf("%d\t",a[i][j]);
  }
  printf("\nThe Second matrix is\n");
  for(i=0;i<3;i++){
      printf("\n");
      for(j=0;j<3;j++)
      printf("%d\t",b[i][j]);
   }
   #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
   {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

     int *dev_a;
     int *dev_b;
     int *dev_c;
     
     cudaCheckReturn(hipMalloc((void **) &dev_a, (3) * (3) * sizeof(int)));
     cudaCheckReturn(hipMalloc((void **) &dev_b, (3) * (3) * sizeof(int)));
     cudaCheckReturn(hipMalloc((void **) &dev_c, (3) * (3) * sizeof(int)));
     
     cudaCheckReturn(hipMemcpy(dev_a, a, (3) * (3) * sizeof(int), hipMemcpyHostToDevice));
     cudaCheckReturn(hipMemcpy(dev_b, b, (3) * (3) * sizeof(int), hipMemcpyHostToDevice));
     {
       dim3 k0_dimBlock(3, 3);
       dim3 k0_dimGrid(1, 1);
       kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_a, dev_b, dev_c);
       cudaCheckKernel();
     }
     
     cudaCheckReturn(hipMemcpy(c, dev_c, (3) * (3) * sizeof(int), hipMemcpyDeviceToHost));
     cudaCheckReturn(hipFree(dev_a));
     cudaCheckReturn(hipFree(dev_b));
     cudaCheckReturn(hipFree(dev_c));
   }
   printf("\nThe Addition of two matrix is\n");
   for(i=0;i<3;i++){
       printf("\n");
       for(j=0;j<3;j++)
            printf("%d\t",c[i][j]);
   }
   return 0;
}
