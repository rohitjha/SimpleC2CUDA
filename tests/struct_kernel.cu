#include "hip/hip_runtime.h"
#include "struct_kernel.hu"
struct s {
    int c[10][10];
};
__global__ void kernel0(struct s *b)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.z, t1 = threadIdx.y, t2 = threadIdx.x;

    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    for (int c5 = t1; c5 <= 9; c5 += 4)
      for (int c6 = t2; c6 <= 9; c6 += 4)
        for (int c7 = 0; c7 <= 9; c7 += 1)
          b[t0 * 10 + c5].c[c6][c7] = ((((t0) + (c5)) + (c6)) + (c7));
}
