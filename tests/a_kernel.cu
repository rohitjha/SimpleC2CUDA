#include "hip/hip_runtime.h"
#include "a_kernel.hu"
__global__ void kernel0(int *a, int *b, int *c)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    int private_a[1];

    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    {
      private_a[0] = a[t0];
      private_a[0] = (private_a[0] + b[t0]);
      c[t0] = private_a[0];
    }
}
