#include "hip/hip_runtime.h"
#include "b_kernel.hu"
__global__ void kernel0(int *a, int *b, int *c)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
    c[t0] = (a[t0] + b[t0]);
}
